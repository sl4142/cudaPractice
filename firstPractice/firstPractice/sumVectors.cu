
#include <hip/hip_runtime.h>
/*#include  "cuda_runtime.h"
#include "device_launch_parameters.h"
#include <stdio.h>


cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main234234() 
{
	const int size = 5;
	const int a[size] = { 1, 2, 3, 4, 5 };
	const int b[size] = { 10, 11 ,12, 13, 14 };
	int c[size] = { 0 };

	cudaError_t cudaStatus = addWithCuda(c, a, b, size);
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "addWithCuda Failed");
		return 1;
	}

	printf("{1,2,3,4,5} + {10,11,12,13,14} = {%d, %d, %d, %d, %d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	cudaStatus = cudaDeviceReset();
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaDeviceReset Failed");
		return 1;
	}

	return 0;
}

cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	cudaError_t cudaStatus;

	cudaStatus = cudaSetDevice(0);
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaSetDevice Failed");
	}
	int newSize = size * sizeof(int);

	cudaMalloc((void**)&dev_a, newSize);
	cudaMalloc((void**)&dev_b, newSize);
	cudaMalloc((void**)&dev_c, newSize);
	cudaMemcpy(dev_a, a, newSize, cudaMemcpyHostToDevice);
	cudaMemcpy(dev_b, b, newSize, cudaMemcpyHostToDevice);
	addKernel<<<1, size >>> (dev_c, dev_a, dev_b);
	cudaDeviceSynchronize();
	cudaMemcpy(c, dev_c, newSize, cudaMemcpyDeviceToHost);

	cudaFree(dev_c);
	cudaFree(dev_a);
	cudaFree(dev_b);

	return cudaStatus;

}
*/