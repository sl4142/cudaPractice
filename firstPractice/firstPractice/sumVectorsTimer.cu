#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

hipError_t cudaAdd(float *c, const float *a, const float *b, unsigned int size);
__global__ void addKernel(float *c, const float *a, const float *b)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	c[i] = a[i] + b[i];
}


void sumWithHost(float *c, const float *a, const float *b, int size)
{
	for (int i = 0; i < size; i++)
	{
		c[i] = a[i] + b[i];
	}
}


void initData(float *a, int size)
{
	time_t t;
	
	srand((unsigned) time(&t));
	for (int i = 0; i < size; i++)
	{
		a[i] = (float)(rand() & 0xFF) / 10.0f;
	}
}

int main()
{
	/*const int size = 5000000;
	float a[size];

	clock_t start, end;
	double timeUsed;
	start = clock();
	fun();
	timeUsed = ((double)(clock() - start) / CLOCKS_PER_SEC);
	printf("HOST CPU Time Used = %.10f\n", timeUsed);*/

	/*const int size = 10;
	int nBytes = size * sizeof(float);
	printf("nBytes = %d\n", nBytes);
	float a[size], b[size], c[size];
	//a = (float *)malloc(nBytes);
	//b = (float *)malloc(nBytes);
	//c = (float *)malloc(nBytes);
	
	initData(a, size);
	initData(b, size);*/
	
	printf("start...\n");

	const unsigned int size = 3000;
	float a[size], b[size], c[size];
	initData(a, size);
	initData(b, size);

	/*clock_t start = clock();
	sumWithHost(c, a, b, size);
	double timeUsed = clock() - start;
	timeUsed = ((double)(timeUsed / CLOCKS_PER_SEC));
	printf("Host CPU Time Used = %.15f\n", timeUsed);
	printf("c %.2f = a %.2f + b %.2f\n", c[size-1], a[size-1], b[size-1]);
	//free(a);*/

	cudaAdd(c, a, b, size);
	printf("c %.2f = a %.2f + b %.2f GPU\n", c[1], a[1], b[1]);
}

hipError_t cudaAdd(float *c, const float *a, const float *b, unsigned int size)
{
	float *dev_c, *dev_a, *dev_b;
	const int newSize = size * sizeof(float);
	hipSetDevice(0);
	hipMalloc((void**)&dev_c, size * sizeof(float));
	hipMalloc((void**)&dev_b, size * sizeof(float));
	hipMalloc((void**)&dev_a, size * sizeof(float));
	hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);
	addKernel << <1, size >> > (dev_c, dev_a, dev_b);
	hipDeviceSynchronize();
	hipMemcpy(c, dev_c, size * sizeof(float), hipMemcpyDeviceToHost);
	hipError_t status = hipDeviceReset();
	hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
	return status;
}
