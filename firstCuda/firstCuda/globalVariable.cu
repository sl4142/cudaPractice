#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>


__device__ float devData;

__global__ void checkGlobalVariable()
{
	printf("Device: %f\n", devData);
	devData += 2.0f;
}


int main(int argc, char **argv)
{

	float value = 3.14f;
	hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float));
	printf("Host copied %f\n", value);
	checkGlobalVariable << <1, 1 >> > ();
	hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float));
	printf("Host changed %f\n", value);

	hipDeviceReset();
	return EXIT_SUCCESS;

	printf("Hello\n");
}