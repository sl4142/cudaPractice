#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

/*
THIS VERSION IS PERFECT!!!
*/

/*void sumOnHost(float *c, const float *a, const float *b, unsigned int size)
{
	for (int i = 0; i < size; i++)
	{
		c[i] = a[i] + b[i];
	}
}

cudaError_t cudaAdd(float *c, const float *a, const float *b, unsigned long size);
__global__ void addKernel(float *c, const float *a, const float *b, unsigned int size)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) c[i] = a[i] + b[i];
}

void initData(float *dat, unsigned long size)
{
	time_t t;
	srand((unsigned)time(&t));
	for (long i = 0; i < size; i++)
	{
		dat[i] = (rand() & 0xFF) / 10.0f;
	}
}

void checkResult(const float *a, const float *b, unsigned long size)
{
	double eps = 1.0E-8;
	for (long i = 0; i < size; i++)
	{
		if (abs(a[i] - b[i]) > eps)
		{
			printf("Not Match!\n");
			break;
		}
	}
	printf("Results Match!\n");

}

int main(int argc, char **argv)
{
	printf("Started...\n");
	const long size = 100000000;
	const long nBytes = size * sizeof(float);
	float *a = (float *)malloc(nBytes);
	float *b = (float *)malloc(nBytes);
	float *c = (float *)malloc(nBytes);
	float *gpuC = (float *)malloc(nBytes);


	initData(a, size);
	initData(b, size);
	
	time_t t = clock();
	for (long i = 0; i < size; i++)
	{
		c[i] = a[i] + b[i]; 
	}
	printf("size = %d\n", size);
	double used = clock() - t;
	used = (double)used / CLOCKS_PER_SEC;
	printf("Host time used = %.10f\n", used);
	printf("%.3f = %.3f + %.3f\n", c[1], a[1], b[1]);

	
	cudaAdd(gpuC, a, b, size);
	printf("%.3f = %.3f + %.3f\n", gpuC[1], a[1], b[1]);

	checkResult(c, gpuC, size);
}

cudaError_t cudaAdd(float *c, const float *a, const float *b, unsigned long size)
{
	const long newSize = size * sizeof(float);
	float *dev_c = 0, *dev_a = 0, *dev_b = 0;
	cudaSetDevice(0);
	cudaMalloc((void**)&dev_c, newSize);
	cudaMalloc((void**)&dev_a, newSize);
	cudaMalloc((void**)&dev_b, newSize);
	cudaMemcpy(dev_a, a, newSize, cudaMemcpyHostToDevice);
	cudaMemcpy(dev_b, b, newSize, cudaMemcpyHostToDevice);
	dim3 block(1024);
	dim3 grid((size + block.x - 1) / block.x);
	time_t t = clock();
	addKernel << <grid, block>> > (dev_c, dev_a, dev_b, size);
	cudaDeviceSynchronize();
	double used = clock() - t;
	used = (double)used / CLOCKS_PER_SEC;
	printf("<<<%d, %d>>> GPU time used = %.10f\n", grid.x, block.x, used);
	cudaMemcpy(c, dev_c, newSize, cudaMemcpyDeviceToHost);
	cudaFree(dev_c);
	cudaFree(dev_b);
	cudaFree(dev_a);
	cudaError_t status = cudaDeviceReset();
	return status;
}*/