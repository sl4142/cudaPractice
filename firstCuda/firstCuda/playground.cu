#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

/*#define CHECK(call)																	\
{																					\
	cudaError_t error = call;														\
	if (error != cudaSuccess)														\
	{																				\
		fprintf(stderr, "%s: %d", __FILE__, __LINE__);								\
		fprintf(stderr, "%code %d, reason: %s", error, cudaGetErrorString(error));	\
		exit(1);																	\
	}																				\
}																					\

void initalData(float *dat, const int size) 
{
	time_t t;
	srand((unsigned)time(&t));
	for (int i = 0; i < size; i++) {
		dat[i] = (rand() & 0xFF) / 10.0f;
	}
}

void checkResult(const float *a, const float *b, const unsigned int size) {
	double eps = 1.0E-08;
	for (int i = 0; i < size; i++) 
	{
		if (abs(a[i] - b[i]) > eps)
		{
			printf("Result Not Match!\n");
			return;
		}
	}
	printf("Results Match!\n");
}

void sumOnCPU(float *c, float *a, float *b, const int nx, const int ny) {
	float *C = c;
	float *A = a;
	float *B = b;
	for (int iy = 0; iy < ny; iy++)
	{
		for (int ix = 0; ix < nx; ix++)
		{
			C[ix] = A[ix] + B[ix];
		}
		C += nx;
		A += nx;
		B += nx;
	}
}

cudaError_t cudaSumMatrix(float *c, const float *a, const float *b, const int nx, const int ny);

__global__ void addVectorKernel(float *c, float *a, float *b, const int size)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size) c[idx] = a[idx] + b[idx];
}

cudaError_t cudaSumVector(float *c, const float *a, const float *b, const int size)
{
	float *dev_c = 0, *dev_a = 0, *dev_b = 0;
	int nBytes = size * sizeof(float);
	cudaMalloc((void**)&dev_c, nBytes);
	cudaMalloc((void**)&dev_c, nBytes);
	cudaMalloc((void**)&dev_c, nBytes);

	cudaMemcpy(dev_a, a, nBytes, cudaMemcpyHostToDevice);
	cudaMemcpy(dev_a, a, nBytes, cudaMemcpyHostToDevice);

	dim3 block(1024);
	dim3 grid((size + block.x - 1) / block.x);
	addVectorKernel << <grid, block >> > (dev_c, dev_a, dev_b, size);
	cudaDeviceSynchronize();
	cudaMemcpy(c, dev_c, nBytes, cudaMemcpyDeviceToHost);
	cudaFree(dev_c);
	cudaFree(dev_a);
	cudaFree(dev_b);
	cudaError_t status = cudaDeviceReset();
	return status;
}

__global__ void addKernel(float *c, const float *a, const float *b, const int nx, const int ny)
{
	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iy = threadIdx.y + blockIdx.y * blockDim.y;
	int idx = iy * nx + ix;
	if (ix < nx && iy < ny) c[idx] = a[idx] + b[idx];
}

int main(int argc, char **argv)
{
	const int nx = 1 << 14, ny = 1 << 14;
	const int nxy = nx * ny;
	const int nBytes = nxy * sizeof(float);
	float *a = (float*)malloc(nBytes);
	float *b = (float*)malloc(nBytes);
	float *c = (float*)malloc(nBytes);
	float *gpuC = (float*)malloc(nBytes);

	printf("Started...\n");
	printf("Matrix %d x %d\n", nx, ny);

	time_t t = clock();
	initalData(a, nxy);
	initalData(b, nxy);
	double used = clock() - t;
	used = (double)used / CLOCKS_PER_SEC;
	printf("Initial Data used = %.3f\n", used);

	t = clock();
	sumOnCPU(c, a, b, nx, ny);
	used = clock() - t;
	used = (double)used / CLOCKS_PER_SEC;
	printf("CPU used = %.3f\n", used);

	cudaSumMatrix(gpuC, a, b, nx, ny);
	checkResult(c, gpuC, nxy);

	return 0;

}

cudaError_t cudaSumMatrix(float *c, const float *a, const float *b, const int nx, const int ny)
{
	float *dev_a = 0, *dev_b = 0, *dev_c = 0;
	cudaDeviceProp devProp;
	int dev = 0;
	cudaGetDeviceProperties(&devProp, dev);
	printf("Device %d: %s\n", dev, devProp.name);
	const int nxy = nx * ny;
	const int nBytes = nxy * sizeof(float);
	cudaSetDevice(dev);
	cudaMalloc((void**)&dev_a, nBytes);
	cudaMalloc((void**)&dev_b, nBytes);
	cudaMalloc((void**)&dev_c, nBytes);

	cudaMemcpy(dev_a, a, nBytes, cudaMemcpyHostToDevice);
	cudaMemcpy(dev_b, b, nBytes, cudaMemcpyHostToDevice);
	int dimx = 32, dimy = 32;
	dim3 block(dimx, dimy);
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
	time_t t = clock();
	addKernel << <grid, block>> > (dev_c, dev_a, dev_b, nx, ny);
	cudaDeviceSynchronize();
	double used = clock() - t;
	used = (double)used / CLOCKS_PER_SEC;
	printf("<<<(%d,%d), (%d,%d)>>> GPU used = %.3f\n", 
		grid.x, grid.y, block.x, block.y, used);
	cudaMemcpy(c, dev_c, nBytes, cudaMemcpyDeviceToHost);
	cudaFree(dev_c);
	cudaFree(dev_a);
	cudaFree(dev_b);
	cudaError_t status = cudaDeviceReset();
	return status;
}*/



