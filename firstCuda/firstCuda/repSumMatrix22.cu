#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

/*void initData(float *c, int size)
{
	time_t t;
	srand((unsigned)time(&t));
	for (int i = 0; i < size; i++)
	{
		c[i] = (rand() & 0xFF) / 10.0f;
	}
}

//sumMatrixOnHost
void sumMatrixOnHost(float *c, float *a, float *b, const int nx, const int ny)
{
	float *A = a;
	float *B = b;
	float *C = c;
	for (int iy = 0; iy < ny; iy++)
	{
		for (int ix = 0; ix < nx; ix++)
		{
			C[ix] = A[ix] + B[ix];
		}

		A += nx;
		B += nx;
		C += nx;
	}
}

// checkResult
void checkResult(const float *a, const float *b, const int size)
{
	double eps = 1.0E-8;
	for (int i = 0; i < size; i++)
	{
		if (abs(a[i] - b[i]) > eps)
		{
			printf("Results Not Match!\n");
			return;
		}
	}
	printf("Resutls Match!\n");
}

//cudaSumMatrix
cudaError_t cudaSumMatrix2D(float *c, const float *a, const float *b, const int nx, const int ny);

//sumKernel
__global__ void sumMatrixKernel(float *c, const float *a, const float *b, const int nx, const int ny)
{
	int ix = threadIdx.x + blockIdx.x * blockDim.x;
	int iy = threadIdx.y + blockIdx.y * blockDim.y;
	int idx = iy * nx + ix;
	if (ix < nx && iy < ny) c[idx] = a[idx] + b[idx];
}

//main
int main(int argc, char **argv)
{
	const int nx = 1 << 14, ny = 1 << 14;
	const int nxy = nx * ny;
	const int nBytes = nxy * sizeof(float);
	float *A = (float*)malloc(nBytes);
	float *B = (float*)malloc(nBytes);
	float *C = (float*)malloc(nBytes);
	float *gpuC = (float*)malloc(nBytes);
	printf("Started...\n");
	printf("Matrix %d x %d\n", nx, ny);
	time_t t = clock();
	initData(A, nxy);
	initData(B, nxy);
	double used = clock() - t;
	used = (double)used / CLOCKS_PER_SEC;
	printf("Initial Data used = %.3f\n", used);

	t = clock();
	sumMatrixOnHost(C, A, B, nx, ny);
	used = clock() - t;
	used = (double)used / CLOCKS_PER_SEC;
	printf("CPU used = %.3f\n", used);

	cudaSumMatrix2D(gpuC, A, B, nx, ny);
	checkResult(C, gpuC, nxy);
	free(A); free(B); free(C); free(gpuC);
	return 0;
}

cudaError_t cudaSumMatrix2D(float *c, const float *a, const float *b, const int nx, const int ny)
{
	cudaDeviceProp devProp;
	int dev = 0;
	cudaGetDeviceProperties(&devProp, dev);
	printf("Device %d: %s\n", dev, devProp.name);
	cudaSetDevice(dev);
	float *dev_a = 0, *dev_b = 0, *dev_c = 0;
	int nBytes = nx * ny * sizeof(float);
	cudaMalloc((void**)&dev_a, nBytes);
	cudaMalloc((void**)&dev_b, nBytes);
	cudaMalloc((void**)&dev_c, nBytes);

	cudaMemcpy(dev_a, a, nBytes, cudaMemcpyHostToDevice);
	cudaMemcpy(dev_b, b, nBytes, cudaMemcpyHostToDevice);

	int dimx = 32, dimy = 32;
	dim3 block(dimx, dimy);
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
	time_t t = clock();
	sumMatrixKernel << <grid, block >> > (dev_c, dev_a, dev_b, nx, ny);
	cudaDeviceSynchronize();
	double used = clock() - t;
	used = (double)used / CLOCKS_PER_SEC;
	printf("GPU used = %.3f\n", used);
	cudaMemcpy(c, dev_c, nBytes, cudaMemcpyDeviceToHost);
	cudaFree(dev_a); cudaFree(dev_b); cudaFree(dev_c);
	cudaError_t status = cudaDeviceReset();
	return status;
}*/








