
#include <hip/hip_runtime.h>

/*#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

cudaError_t addWithCuda(long *c, const long *a, const long *b, unsigned long size);

__global__ void addKernel(long *c, const long *a, const long *b)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const long arraySize = 100000;
	long a[arraySize];
	long b[arraySize];
	long *c = (long *)malloc(arraySize * sizeof(long));
	long *cpuC = (long *)malloc(arraySize * sizeof(long));
	for (long i = 0; i < arraySize; i++)
	{
		a[i] = i+1;
		b[i] = (i+1);
	}

	time_t t = clock();
	for (long i = 0; i < arraySize; i++)
	{
		c[i] = a[i] + b[i];
	}
	double used = clock() - t;
	used = (double)used / CLOCKS_PER_SEC;
	printf("CPU Time = %.10f\n", used);


    // Add vectors in parallel.
    cudaError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{%d,%d,%d,%d,%d}\n",
        c[arraySize-5], c[arraySize-4], c[arraySize-3], c[arraySize-2], c[arraySize-1]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = cudaDeviceReset();
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
cudaError_t addWithCuda(long *c, const long *a, const long *b, unsigned long size)
{
	long *dev_a = 0;
	long *dev_b = 0;
	long *dev_c = 0;
    cudaError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = cudaSetDevice(0);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = cudaMalloc((void**)&dev_c, size * sizeof(long));
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMalloc failed!");
        goto Error;
    }

    cudaStatus = cudaMalloc((void**)&dev_a, size * sizeof(long));
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMalloc failed!");
        goto Error;
    }

    cudaStatus = cudaMalloc((void**)&dev_b, size * sizeof(long));
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = cudaMemcpy(dev_a, a, size * sizeof(long), cudaMemcpyHostToDevice);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMemcpy failed!");
        goto Error;
    }

    cudaStatus = cudaMemcpy(dev_b, b, size * sizeof(long), cudaMemcpyHostToDevice);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMemcpy failed!");
        goto Error;
    }

	dim3 block(1024);
	dim3 grid((size + block.x - 1) / block.x);

	time_t t = clock();

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<grid, block>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = cudaGetLastError();
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = cudaDeviceSynchronize();
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

	double used = clock() - t;
	used = (double)used / CLOCKS_PER_SEC;
	printf("<<< %d, %d>>> GPU Time = %.10f\n", grid.x, block.x, used);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = cudaMemcpy(c, dev_c, size * sizeof(long), cudaMemcpyDeviceToHost);
    if (cudaStatus != cudaSuccess) {
        fprintf(stderr, "cudaMemcpy failed!");
        goto Error;
    }

Error:
    cudaFree(dev_c);
    cudaFree(dev_a);
    cudaFree(dev_b);
    
    return cudaStatus;
}
*/